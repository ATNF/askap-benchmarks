#include "hip/hip_runtime.h"
#include "WarmupGPU.h"

using std::vector;
using std::cout;
using std::endl;

__global__
void vectorAdd(const float* a, const float* b, float* c, const size_t N)
{
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
	{
		for (int j = 0; j < 250; ++j)
		{
			c[i] = a[i] + b[i];
		}
	}
}

void WarmupGPU::warmup() const
{
	vector<float> a(N, 1.0);
	vector<float> b(N, 2.0);
	vector<float> c(N, 0.0);
	vector<float> cAnswer(N, 3.0);

	const size_t SIZE = N * sizeof(float);

	float* dA;
	float* dB;
	float* dC;

	hipMalloc(&dA, SIZE);
	hipMalloc(&dB, SIZE);
	hipMalloc(&dC, SIZE);

	hipMemcpy(dA, a.data(), SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dB, b.data(), SIZE, hipMemcpyHostToDevice);

	const int blockSize = 1024;
	const int gridSize = N / 1024;

	vectorAdd <<<gridSize, blockSize>>> (dA, dB, dC, N);

	hipMemcpy(c.data(), dC, SIZE, hipMemcpyDeviceToHost);

	MaxError<float> maximumError;
	cout << "Verifying warmup launch" << endl;
	maximumError.maxError(c, cAnswer);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}
