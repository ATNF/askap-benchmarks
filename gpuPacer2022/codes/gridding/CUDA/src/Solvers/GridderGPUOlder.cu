#include "hip/hip_runtime.h"
#include "GridderGPUOlder.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

__global__
void devGridKernelOlder(
    const Complex * data,
    const int support,
    const Complex * C,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex * grid,
    const int GSIZE,
    const int dind)
{
    // The actual starting grid point
    __shared__ int gindShared;

    // The Convolution function point from which we offset
    __shared__ int cindShared;

    // Calculate the data index offset for this block
    const int dindLocal = dind + blockIdx.y;

    // A copy of the visibility data
    // All threads can read it from shared memory
    // rather than all reading from device (global) memory
    __shared__ Complex dataLocal;

    if (threadIdx.x == 0)
    {
        gindShared = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
        cindShared = cOffset[dindLocal];
        dataLocal = data[dindLocal];
    }

    __syncthreads();

    // Make a local copy from shared memory
    int gind = gindShared;
    int cind = cindShared;

    // blockIdx.x gives the support location in the v direction
    int SSIZE = 2 * support + 1;
    gind += GSIZE * blockIdx.x;
    cind += SSIZE * blockIdx.x;

    // threadIdx.x gives the support location in the u direction
    grid[gind + threadIdx.x] = hipCfmaf(dataLocal, C[cind + threadIdx.x], grid[gind + threadIdx.x]);
}

template<typename T2>
int GridderGPUOlder<T2>::gridStep(const int DSIZE, const int SSIZE, const int dind)
{
    const int MAXSAMPLES = 32;
    for (int step = 1; step <= MAXSAMPLES; ++step)
    {
        for (int check = (step - 1); check >= 0; --check)
        {
            if (!((dind + step) < DSIZE && (
                abs(this->iu[dind + step] - this->iu[dind + check]) > SSIZE ||
                abs(this->iv[dind + step] - this->iv[dind + check]) > SSIZE)))
            {
                return step;
            }
        }
    }
    return MAXSAMPLES;
}

template<typename T2>
void GridderGPUOlder<T2>::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");
}

template<typename T2>
void GridderGPUOlder<T2>::copyH2D()
{
    hipMemcpy(dData, this->data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, this->grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, this->C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, this->cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, this->iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, this->iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
}

template<typename T2>
GridderGPUOlder<T2>::~GridderGPUOlder()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
}

template <typename T2>
void GridderGPUOlder<T2>::gridder()
{
    cout << "\nGridding on GPU" << endl;
    deviceAllocations();
    copyH2D();

    // Kernel launch
    cout << "Kernel launch" << endl;
    const size_t DSIZE = this->data.size();
    typedef hipComplex Complex;

    const int SSIZE = 2 * this->support + 1;

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), hipFuncCachePreferL1);

    int step = 1;
    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += step)
    {
        step = gridStep(DSIZE, SSIZE, dind);
        dim3 gridDim(SSIZE, step);
        /// PJE: make sure any chevron is tightly packed
        devGridKernelOlder << <gridDim, SSIZE >> > ((const Complex*)dData, this->support, (const Complex*)dC,
            dCOffset, dIU, dIV, (Complex*)dGrid, GSIZE, dind);
        cudaCheckErrors("kernel launch (devGridKernel_v0) failure");
        count++;
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(this->grid.data(), dGrid, SIZE_GRID, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
}

template void GridderGPUOlder<std::complex<float>>::gridder();
template void GridderGPUOlder<std::complex<double>>::gridder();
template void GridderGPUOlder<std::complex<float>>::deviceAllocations();
template void GridderGPUOlder<std::complex<double>>::deviceAllocations();
template void GridderGPUOlder<std::complex<float>>::copyH2D();
template void GridderGPUOlder<std::complex<double>>::copyH2D();
template GridderGPUOlder<std::complex<float>>::~GridderGPUOlder();
template GridderGPUOlder<std::complex<double>>::~GridderGPUOlder();
template int GridderGPUOlder<std::complex<float>>::gridStep(const int DSIZE, const int SSIZE, const int dind);
template int GridderGPUOlder<std::complex<double>>::gridStep(const int DSIZE, const int SSIZE, const int dind);
