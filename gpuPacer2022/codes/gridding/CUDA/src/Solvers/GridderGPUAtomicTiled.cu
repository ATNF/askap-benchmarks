#include "hip/hip_runtime.h"
#include "GridderGPUAtomicTiled.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

__global__
void devGridKernelAtomicTiled(
    const Complex* data,
    const int support,
    const Complex* C,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* grid,
    const int GSIZE,
    const int i)
{
    const int SSIZE = 2 * support + 1;

    const int tID = threadIdx.x;
    const int dind = i + blockIdx.x * blockDim.x + threadIdx.x;

    // The actual starting grid point
    int gind = iu[dind] + GSIZE * iv[dind] - support;
    // The Convolution function point from which we offset
    int cind = cOffset[dind];

    Complex dataLocal = data[dind];

    __shared__ int suppU;
    __shared__ int suppV;

    if (tID == 0)
    {
        suppU = blockIdx.y;
        suppV = blockIdx.z;
    }
    __syncthreads();

    // blockIdx.z gives the support location in the v direction
    gind += GSIZE * suppV;
    cind += SSIZE * suppV;


    //Complex gLocal = hipCfmaf(dataLocal, C[cind + suppU], grid[gind + suppU]);
    //grid[gind + suppU] = hipCfmaf(dataLocal, C[cind + suppU], grid[gind + suppU]);
    //atomicAdd(&grid[gind + suppU].x, gLocal.x);
    //atomicAdd(&grid[gind + suppU].y, gLocal.y);
    //grid[gind + suppU].x += gLocal.x;
    atomicAdd(&grid[gind + suppU].x, dataLocal.x * C[cind + suppU].x - dataLocal.y * C[cind + suppU].y);
    atomicAdd(&grid[gind + suppU].y, dataLocal.x * C[cind + suppU].y + dataLocal.y * C[cind + suppU].x);

    //grid[gind + suppU] = hipCfmaf(dataLocal, C[cind + suppU], grid[gind + suppU]);
}

void GridderGPUAtomicTiled::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");
}

void GridderGPUAtomicTiled::copyH2D()
{
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
}

GridderGPUAtomicTiled::~GridderGPUAtomic()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
}

void GridderGPUAtomicTiled::gridder()
{
    cout << "\nGridding on GPU" << endl;
    deviceAllocations();
    copyH2D();

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE_Y = 129;
    const int GRID_SIZE_Z = 129;
    const int GRID_SIZE_X = NSAMPLES / BLOCK_SIZE;

    // Kernel launch
    cout << "Kernel launch" << endl;
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelAtomicTiled), hipFuncCachePreferL1);

    dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z);
    int stepSize = GRID_SIZE_X * BLOCK_SIZE;

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += stepSize)
    {

        ++count;

        devGridKernelAtomicTiled <<< gridSize, BLOCK_SIZE >>> ((const Complex*)dData, support, (const Complex*)dC,
            dCOffset, dIU, dIV, (Complex*)dGrid, GSIZE, dind);

        cudaCheckErrors("cuda kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(grid.data(), dGrid, SIZE_GRID, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
}
