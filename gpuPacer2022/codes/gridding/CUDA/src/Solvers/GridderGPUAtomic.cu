#include "hip/hip_runtime.h"
#include "GridderGPUAtomic.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

__global__
void devGridKernelAtomic(
        const Complex* data,
        const int support,
        const Complex* C,
        const int* cOffset,
        const int* iu,
        const int* iv,
        Complex* grid,
        const int GSIZE,
        const int dind)
{

    const int SSIZE = 2 * support + 1;
    assert(SSIZE == blockDim.x);

    const int bind = blockIdx.x;
    const int tind = threadIdx.x;

    const int dindLocal = dind + bind;

    int gind = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
    int cind = cOffset[dindLocal];
    const Complex dataLocal = data[dindLocal];

    for (int row = 0; row < SSIZE; ++row)
    {

        if (tind < SSIZE)
        {
            //grid[gind + tind] = hipCfmaf(dataLocal, C[cind + tind], grid[gind + tind]);
            const Complex tmp = hipCmulf(dataLocal, C[cind + tind]);
            //grid[gind + tind] = hipCaddf(grid[gind + tind], tmp);
            atomicAdd(&grid[gind].x + 2 * tind, tmp.x);
            atomicAdd(&grid[gind].y + 2 * tind + 1, tmp.y);
        }

        gind += GSIZE;
        cind += SSIZE;

    }
}

template<typename T2>
void GridderGPUAtomic<T2>::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");
}

template<typename T2>
void GridderGPUAtomic<T2>::copyH2D()
{
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
}

template<typename T2>
GridderGPUAtomic<T2>::~GridderGPUAtomic()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
}

template <typename T2>
void GridderGPUAtomic<T2>::gridder()
{
    cout << "\nGridding on GPU" << endl;
    deviceAllocations();
    copyH2D();

    // Kernel launch
    cout << "Kernel launch" << endl;
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;

    const int SSIZE = 2 * support + 1;

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelAtomic), hipFuncCachePreferL1);

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devGridKernelAtomic <<< gridSize, SSIZE >>> ((const Complex*)dData, support, (const Complex*)dC,
            dCOffset, dIU, dIV, (Complex*)dGrid, GSIZE, dind);

        cudaCheckErrors("cuda kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(grid.data(), dGrid, SIZE_GRID, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
}

template void GridderGPUAtomic<std::complex<float>>::gridder();
template void GridderGPUAtomic<std::complex<double>>::gridder();
template void GridderGPUAtomic<std::complex<float>>::deviceAllocations();
template void GridderGPUAtomic<std::complex<double>>::deviceAllocations();
template void GridderGPUAtomic<std::complex<float>>::copyH2D();
template void GridderGPUAtomic<std::complex<double>>::copyH2D();
template GridderGPUAtomic<std::complex<float>>::~GridderGPUAtomic();
template GridderGPUAtomic<std::complex<double>>::~GridderGPUAtomic();