#include "hip/hip_runtime.h"
#include "GridderGPUOlder.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

__global__
void devGridKernelOlder(
    const Complex * data,
    const int support,
    const Complex * C,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex * grid,
    const int GSIZE,
    const int dind)
{
    // The actual starting grid point
    __shared__ int gindShared;

    // The Convolution function point from which we offset
    __shared__ int cindShared;

    // Calculate the data index offset for this block
    const int dindLocal = dind + blockIdx.y;

    // A copy of the visibility data
    // All threads can read it from shared memory
    // rather than all reading from device (global) memory
    __shared__ Complex dataLocal;

    if (threadIdx.x == 0)
    {
        gindShared = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
        cindShared = cOffset[dindLocal];
        dataLocal = data[dindLocal];
    }

    __syncthreads();

    // Make a local copy from shared memory
    int gind = gindShared;
    int cind = cindShared;

    // blockIdx.x gives the support location in the v direction
    int SSIZE = 2 * support + 1;
    gind += GSIZE * blockIdx.x;
    cind += SSIZE * blockIdx.x;

    // threadIdx.x gives the support location in the u direction
    grid[gind + threadIdx.x] = gpuCfmaf(dataLocal, C[cind + threadIdx.x], grid[gind + threadIdx.x]);
}

template<typename T2>
int GridderGPUOlder<T2>::gridStep(const int DSIZE, const int SSIZE, const int dind)
{
    const int MAXSAMPLES = 32;
    for (int step = 1; step <= MAXSAMPLES; ++step)
    {
        for (int check = (step - 1); check >= 0; --check)
        {
            if (!((dind + step) < DSIZE && (
                abs(this->iu[dind + step] - this->iu[dind + check]) > SSIZE ||
                abs(this->iv[dind + step] - this->iv[dind + check]) > SSIZE)))
            {
                return step;
            }
        }
    }
    return MAXSAMPLES;
}

template<typename T2>
void GridderGPUOlder<T2>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dData, SIZE_DATA);
    gpuMalloc(&dGrid, SIZE_GRID);
    gpuMalloc(&dC, SIZE_C);
    gpuMalloc(&dCOffset, SIZE_COFFSET);
    gpuMalloc(&dIU, SIZE_IU);
    gpuMalloc(&dIV, SIZE_IV);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T2>
void GridderGPUOlder<T2>::copyH2D()
{
    gpuMemcpy(dData, this->data.data(), SIZE_DATA, gpuMemcpyHostToDevice);
    gpuMemcpy(dGrid, this->grid.data(), SIZE_GRID, gpuMemcpyHostToDevice);
    gpuMemcpy(dC, this->C.data(), SIZE_C, gpuMemcpyHostToDevice);
    gpuMemcpy(dCOffset, this->cOffset.data(), SIZE_COFFSET, gpuMemcpyHostToDevice);
    gpuMemcpy(dIU, this->iu.data(), SIZE_IU, gpuMemcpyHostToDevice);
    gpuMemcpy(dIV, this->iv.data(), SIZE_IV, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T2>
GridderGPUOlder<T2>::~GridderGPUOlder()
{
    // Deallocate device vectors
    gpuFree(dData);
    gpuFree(dGrid);
    gpuFree(dC);
    gpuFree(dCOffset);
    gpuFree(dIU);
    gpuFree(dIV);
    gpuCheckErrors("gpuFree failure");
}

template <typename T2>
void GridderGPUOlder<T2>::gridder()
{
    cout << "\nGridding on GPU" << endl;
    deviceAllocations();
    copyH2D();

    // Kernel launch
    cout << "Kernel launch" << endl;
    const size_t DSIZE = this->data.size();

    const int SSIZE = 2 * this->support + 1;

    gpuFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), gpuFuncCachePreferL1);

    int step = 1;
    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += step)
    {
        step = gridStep(DSIZE, SSIZE, dind);
        dim3 gridDim(SSIZE, step);
        /// PJE: make sure any chevron is tightly packed
        devGridKernelOlder <<<gridDim, SSIZE >>> ((const Complex*)dData, this->support, (const Complex*)dC,
            dCOffset, dIU, dIV, (Complex*)dGrid, GSIZE, dind);
        gpuCheckErrors("kernel launch (devGridKernel_v0) failure");
        count++;
    }
    cout << "Used " << count << " kernel launches." << endl;

    gpuMemcpy(this->grid.data(), dGrid, SIZE_GRID, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template void GridderGPUOlder<std::complex<float>>::gridder();
template void GridderGPUOlder<std::complex<double>>::gridder();
template void GridderGPUOlder<std::complex<float>>::deviceAllocations();
template void GridderGPUOlder<std::complex<double>>::deviceAllocations();
template void GridderGPUOlder<std::complex<float>>::copyH2D();
template void GridderGPUOlder<std::complex<double>>::copyH2D();
template GridderGPUOlder<std::complex<float>>::~GridderGPUOlder();
template GridderGPUOlder<std::complex<double>>::~GridderGPUOlder();
template int GridderGPUOlder<std::complex<float>>::gridStep(const int DSIZE, const int SSIZE, const int dind);
template int GridderGPUOlder<std::complex<double>>::gridStep(const int DSIZE, const int SSIZE, const int dind);
