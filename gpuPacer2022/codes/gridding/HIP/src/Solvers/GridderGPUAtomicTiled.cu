#include "hip/hip_runtime.h"
#include "GridderGPUAtomicTiled.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

__global__
void devGridKernelAtomicTiled(
    const Complex* data,
    const int support,
    const Complex* C,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* grid,
    const int GSIZE,
    const int i)
{
    const int SSIZE = 2 * support + 1;

    const int tID = threadIdx.x;
    const int dind = i + blockIdx.x * blockDim.x + threadIdx.x;

    // The actual starting grid point
    int gind = iu[dind] + GSIZE * iv[dind] - support;
    // The Convolution function point from which we offset
    int cind = cOffset[dind];

    Complex dataLocal = data[dind];

    __shared__ int suppU;
    __shared__ int suppV;

    if (tID == 0)
    {
        suppU = blockIdx.y;
        suppV = blockIdx.z;
    }
    __syncthreads();

    // blockIdx.z gives the support location in the v direction
    gind += GSIZE * suppV;
    cind += SSIZE * suppV;


    //Complex gLocal = hipCfmaf(dataLocal, C[cind + suppU], grid[gind + suppU]);
    //grid[gind + suppU] = hipCfmaf(dataLocal, C[cind + suppU], grid[gind + suppU]);
    //atomicAdd(&grid[gind + suppU].x, gLocal.x);
    //atomicAdd(&grid[gind + suppU].y, gLocal.y);
    //grid[gind + suppU].x += gLocal.x;
    atomicAdd(&grid[gind + suppU].x, dataLocal.x * C[cind + suppU].x - dataLocal.y * C[cind + suppU].y);
    atomicAdd(&grid[gind + suppU].y, dataLocal.x * C[cind + suppU].y + dataLocal.y * C[cind + suppU].x);

    //grid[gind + suppU] = hipCfmaf(dataLocal, C[cind + suppU], grid[gind + suppU]);
}

void GridderGPUAtomicTiled::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    gpuCheckErrors("hipMalloc failure");
}

void GridderGPUAtomicTiled::copyH2D()
{
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    gpuCheckErrors("hipMemcpy H2D failure");
}

GridderGPUAtomicTiled::~GridderGPUAtomic()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    gpuCheckErrors("hipFree failure");
}

void GridderGPUAtomicTiled::gridder()
{
    cout << "\nGridding on GPU" << endl;
    deviceAllocations();
    copyH2D();

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE_Y = 129;
    const int GRID_SIZE_Z = 129;
    const int GRID_SIZE_X = NSAMPLES / BLOCK_SIZE;

    // Kernel launch
    cout << "Kernel launch" << endl;
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelAtomicTiled), hipFuncCachePreferL1);

    dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y, GRID_SIZE_Z);
    int stepSize = GRID_SIZE_X * BLOCK_SIZE;

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += stepSize)
    {

        ++count;

        devGridKernelAtomicTiled <<<gridSize, BLOCK_SIZE>>> ((const Complex*)dData, support, (const Complex*)dC,
            dCOffset, dIU, dIV, (Complex*)dGrid, GSIZE, dind);

        gpuCheckErrors("hip kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(grid.data(), dGrid, SIZE_GRID, hipMemcpyDeviceToHost);
    gpuCheckErrors("hipMemcpy D2H failure");
}
