#include "hip/hip_runtime.h"
#include "GridderGPUAtomic.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

__global__
void devGridKernelAtomic(
        const Complex* data,
        const int support,
        const Complex* C,
        const int* cOffset,
        const int* iu,
        const int* iv,
        Complex* grid,
        const int GSIZE,
        const int dind)
{

    const int SSIZE = 2 * support + 1;
    assert(SSIZE == blockDim.x);

    const int bind = blockIdx.x;
    const int tind = threadIdx.x;

    const int dindLocal = dind + bind;

    int gind = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
    int cind = cOffset[dindLocal];
    const Complex dataLocal = data[dindLocal];

    for (int row = 0; row < SSIZE; ++row)
    {

        if (tind < SSIZE)
        {
            //grid[gind + tind] = hipCfmaf(dataLocal, C[cind + tind], grid[gind + tind]);
            const Complex tmp = hipCmulf(dataLocal, C[cind + tind]);
            //grid[gind + tind] = hipCaddf(grid[gind + tind], tmp);
            atomicAdd(&grid[gind].x + 2 * tind, tmp.x);
            atomicAdd(&grid[gind].y + 2 * tind + 1, tmp.y);
        }

        gind += GSIZE;
        cind += SSIZE;

    }
}

template<typename T2>
void GridderGPUAtomic<T2>::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    gpuCheckErrors("hipMalloc failure");
}

template<typename T2>
void GridderGPUAtomic<T2>::copyH2D()
{
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    gpuCheckErrors("hipMemcpy H2D failure");
}

template<typename T2>
GridderGPUAtomic<T2>::~GridderGPUAtomic()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    gpuCheckErrors("hipFree failure");
}

template <typename T2>
void GridderGPUAtomic<T2>::gridder()
{
    cout << "\nGridding on GPU" << endl;
    deviceAllocations();
    copyH2D();

    // Kernel launch
    cout << "Kernel launch" << endl;
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;

    const int SSIZE = 2 * support + 1;

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelAtomic), hipFuncCachePreferL1);

    int device;
    hipGetDevice(&device);
    hipDeviceProp devProp;
    hipGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devGridKernelAtomic <<<gridSize, SSIZE>>> ((const Complex*)dData, support, (const Complex*)dC,
            dCOffset, dIU, dIV, (Complex*)dGrid, GSIZE, dind);

        gpuCheckErrors("hip kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(grid.data(), dGrid, SIZE_GRID, hipMemcpyDeviceToHost);
    gpuCheckErrors("hipMemcpy D2H failure");
}

template void GridderGPUAtomic<std::complex<float>>::gridder();
template void GridderGPUAtomic<std::complex<double>>::gridder();
template void GridderGPUAtomic<std::complex<float>>::deviceAllocations();
template void GridderGPUAtomic<std::complex<double>>::deviceAllocations();
template void GridderGPUAtomic<std::complex<float>>::copyH2D();
template void GridderGPUAtomic<std::complex<double>>::copyH2D();
template GridderGPUAtomic<std::complex<float>>::~GridderGPUAtomic();
template GridderGPUAtomic<std::complex<double>>::~GridderGPUAtomic();