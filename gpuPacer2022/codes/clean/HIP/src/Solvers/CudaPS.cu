#include "hip/hip_runtime.h"
#include "CudaPS.h"

using std::vector;
using std::cout;
using std::endl;
using std::min;
using std::max;

__global__
void dFindPeak_Step2(float* data, size_t* inIndex, size_t* outIndex, size_t n)
{
    __shared__ float TILE_VAL[BLOCK_SIZE];
    __shared__ size_t TILE_IDX[BLOCK_SIZE];

    size_t tileIdx = threadIdx.x;
    TILE_VAL[tileIdx] = 0.0;
    TILE_IDX[tileIdx] = 0;
    size_t globalIdx = threadIdx.x + blockIdx.x * blockDim.x;

    // grid stride loop to load data
    while (globalIdx < n)
    {
        if (fabs(data[globalIdx]) > fabs(TILE_VAL[tileIdx]))
        {
            TILE_VAL[tileIdx] = data[globalIdx];
            TILE_IDX[tileIdx] = inIndex[globalIdx];
        }
        globalIdx += gridDim.x * blockDim.x;
    }

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();

        // parallel sweep reduction
        if (tileIdx < s)
        {
            if (fabs(TILE_VAL[tileIdx + s]) > fabs(TILE_VAL[tileIdx]))
            {
                TILE_VAL[tileIdx] = TILE_VAL[tileIdx + s];
                TILE_IDX[tileIdx] = TILE_IDX[tileIdx + s];
            }
        }
    }

    if (tileIdx == 0)
    {
        outIndex[blockIdx.x] = TILE_IDX[tileIdx];
        data[blockIdx.x] = TILE_VAL[tileIdx];
    }
}

__global__
void dFindPeak_Step1(const float* data, float* outMax, size_t* outIndex, size_t n)
{
    __shared__ float TILE_VAL[BLOCK_SIZE];
    __shared__ size_t TILE_IDX[BLOCK_SIZE];

    size_t tileIdx = threadIdx.x;

    TILE_VAL[tileIdx] = 0.0;
    TILE_IDX[tileIdx] = 0;

    size_t globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t gridSize = gridDim.x * blockDim.x;

    // grid stride loop to load data
    while (globalIdx < n)
    {
        if (fabs(data[globalIdx]) > fabs(TILE_VAL[tileIdx]))
        {
            TILE_VAL[tileIdx] = data[globalIdx];
            TILE_IDX[tileIdx] = globalIdx;
        }
        globalIdx += gridSize;
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        //__syncthreads();

        // parallel sweep reduction
        if (tileIdx < s)
        {
            if (fabs(TILE_VAL[tileIdx + s]) > fabs(TILE_VAL[tileIdx]))
            {
                TILE_VAL[tileIdx] = TILE_VAL[tileIdx + s];
                TILE_IDX[tileIdx] = TILE_IDX[tileIdx + s];
            }
        }
        __syncthreads();
    }

    if (tileIdx == 0)
    {
        outMax[blockIdx.x] = TILE_VAL[tileIdx];
        outIndex[blockIdx.x] = TILE_IDX[tileIdx];
    }
}

__host__
void CudaPS::reportDevice()
{
    // Report the type of device being used
    int device;
    hipDeviceProp_t devprop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&devprop, device);
    std::cout << "    Using CUDA Device " << device << ": "
        << devprop.name << std::endl;
}

__host__ __device__
CudaPS::Position CudaPS::idxToPos(const size_t idx, const int width)
{
    const int y = idx / width;
    const int x = idx % width;
    return CudaPS::Position(x, y);
}

__host__ __device__
size_t CudaPS::posToIdx(const int width, const CudaPS::Position& pos)
{
    return (pos.y * width) + pos.x;
}

__global__
void dSubtractPSF(const float* dPsf,
    float* dResidual,
    const int imageWidth,
    const int startx, const int starty,
    int const stopx, const int stopy,
    const int diffx, const int diffy,
    const float absPeakVal, const float gain)
{
    const int x = startx + threadIdx.x + (blockIdx.x * blockDim.x);
    const int y = starty + threadIdx.y + (blockIdx.y * blockDim.y);

    // Because thread blocks are of size 16, and the workload is not always
    // a multiple of 16, need to ensure only those threads whose responsibility
    // lies in the work area actually do work
    if (x <= stopx && y <= stopy)
    {
        dResidual[CudaPS::posToIdx(imageWidth, CudaPS::Position(x, y))] -= gain * absPeakVal
            * dPsf[CudaPS::posToIdx(imageWidth, CudaPS::Position(x - diffx, y - diffy))];
    }
}

__host__
void CudaPS::subtractPSF(const size_t peakPos,
    const size_t psfPeakPos,
    const float absPeakVal)
{
    const int blockDim = 16;

    const int rx = idxToPos(peakPos, imageWidth).x;
    const int ry = idxToPos(peakPos, imageWidth).y;

    const int px = idxToPos(psfPeakPos, imageWidth).x;
    const int py = idxToPos(psfPeakPos, imageWidth).y;

    const int diffx = rx - px;
    const int diffy = ry - px;

    const int startx = max(0, rx - px);
    const int starty = max(0, ry - py);

    const int stopx = min(imageWidth - 1, rx + (imageWidth - px - 1));
    const int stopy = min(imageWidth - 1, ry + (imageWidth - py - 1));

    // Note: Both start* and stop* locations are inclusive.
    const int blocksx = ceil((stopx - startx + 1.0) / static_cast<float>(blockDim));
    const int blocksy = ceil((stopy - starty + 1.0) / static_cast<float>(blockDim));

    dim3 numBlocks(blocksx, blocksy);
    dim3 threadsPerBlock(blockDim, blockDim);
    dSubtractPSF << <numBlocks, threadsPerBlock >> > (dPsf, dResidual, imageWidth,
        startx, starty, stopx, stopy, diffx, diffy, absPeakVal, gGain);
    gpuCheckErrors("kernel launch failure in subtractPSF");
}

void CudaPS::deconvolve()
{
    reportDevice();

    residual = dirty;

    // Allocate memory for device vectors
    memAlloc();

    // Copy data from host to device
    copyH2D();

    // Find peak of psf
    Peak psfPeak = findPeak(dPsf, psf.size());

    cout << "Found peak of PSF: " << "Maximum = " << psfPeak.val
        << " at location " << idxToPos(psfPeak.pos, imageWidth).x << ","
        << idxToPos(psfPeak.pos, imageWidth).y << endl;

    for (unsigned int i = 0; i < gNiters; ++i)
    {
        // Find peak in the residual image
        Peak peak = findPeak(dResidual, residual.size());
        if ((i + 1) % 100 == 0 || i == 0)
        {
            cout << "Iteration: " << i + 1 << " - Maximum = " << peak.val
                << " at location " << idxToPos(peak.pos, imageWidth).x << ","
                << idxToPos(peak.pos, imageWidth).y << ", index = " << peak.pos << endl;
        }

        // Check if threshold has been reached
        if (abs(peak.val) < gThreshold)
        {
            cout << "Reached stopping threshold" << endl;
            break;
        }

        // Subtract the PSF from the residual image
        // This function will launch a kernel
        // asynchronously, need to sync later
        subtractPSF(peak.pos, psfPeak.pos, peak.val);
        // Add to model
        model[peak.pos] += peak.val * gGain;
    }

    copyD2H();

}

__host__
CudaPS::Peak CudaPS::findPeak(const float* dData, size_t N)
{
    const size_t SIZE_DATA = N * sizeof(float);
    const size_t SIZE_MAX_VALUE = GRID_SIZE * sizeof(float);
    const size_t SIZE_MAX_INDEX = GRID_SIZE * sizeof(size_t);

    // Host vector for max values
    vector<float> hMax(GRID_SIZE, 0.0);
    // Host vector for index values
    vector<size_t> hIndex(GRID_SIZE, 0);

    // Device vectors
    float* dMax;
    size_t* dIndex;
    size_t* d2Index;

    hipMalloc(&dMax, SIZE_MAX_VALUE);
    hipMalloc(&dIndex, SIZE_MAX_INDEX);
    hipMalloc(&d2Index, sizeof(size_t));
    gpuCheckErrors("hipMalloc failure!");

    dFindPeak_Step1 << <GRID_SIZE, BLOCK_SIZE >> > (dData, dMax, dIndex, N);
    gpuCheckErrors("cuda kernel launch 1 failure!");
    dFindPeak_Step2 << <1, BLOCK_SIZE >> > (dMax, dIndex, d2Index, GRID_SIZE);
    gpuCheckErrors("cuda kernel launch 2 failure!");

    hipMemcpy(hMax.data(), dMax, sizeof(float), hipMemcpyDeviceToHost);
    gpuCheckErrors("hipMemcpy D2H failure in findPeak (hmax)!");
    hipMemcpy(hIndex.data(), d2Index, sizeof(size_t), hipMemcpyDeviceToHost);
    gpuCheckErrors("hipMemcpy D2H failure in findPeak (hindex)!");

    Peak p;
    p.val = hMax[0];
    p.pos = hIndex[0];


    hipFree(dMax);
    hipFree(dIndex);
    hipFree(d2Index);
    gpuCheckErrors("hipFree failure!");

    return p;
}

void CudaPS::memAlloc()
{
    hipMalloc(&dDirty, SIZE_IMAGE);
    hipMalloc(&dPsf, SIZE_IMAGE);
    hipMalloc(&dResidual, SIZE_IMAGE);
    gpuCheckErrors("hipMalloc failure");
}

CudaPS::~CudaPS()
{
    hipFree(dDirty);
    hipFree(dPsf);
    hipFree(dResidual);
    gpuCheckErrors("hipFree failure");
    cout << "Cuda PS destructor" << endl;
}

void CudaPS::copyH2D()
{
    hipMemcpy(dDirty, dirty.data(), SIZE_IMAGE, hipMemcpyHostToDevice);
    hipMemcpy(dPsf, psf.data(), SIZE_IMAGE, hipMemcpyHostToDevice);
    hipMemcpy(dResidual, residual.data(), SIZE_IMAGE, hipMemcpyHostToDevice);
    gpuCheckErrors("hipMemcpy H2D failure");
}

void CudaPS::copyD2H()
{
    hipMemcpy(residual.data(), dResidual, SIZE_IMAGE, hipMemcpyDeviceToHost);
    gpuCheckErrors("hipMemcpy D2H failure");
}
