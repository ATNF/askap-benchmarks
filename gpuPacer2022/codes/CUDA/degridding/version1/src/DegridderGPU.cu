#include "DegridderGPU.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

// Error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

void degridHelper(const Complex* dGrid,
    const int SSIZE,
    const int DSIZE,
    const int GSIZE,
    const int support,
    const Complex* dC,
    const int* dCOffset,
    const int* dIU,
    const int* dIV,
    Complex* dData)
{
    int device;
    hipGetDevice(&device);
    cudaDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    //cout << "maxGridSize "<<devProp.maxGridSize[0]<<" maxThreadsPerBlock = "<<devProp.maxThreadsPerBlock << endl;
    int gridSize = devProp.maxGridSize[0]/(support+1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devDegridKernel <<< gridSize, SSIZE >>>(dGrid, GSIZE, dC, support, dCOffset, dIU, dIV, dData, dind);

        cudaCheckErrors("cuda kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

}

template <typename T2>
void DegridderGPU<T2>::degridder()
{
    cout << "\nDegridding on GPU" << endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float tAlloc{ 0.0 }; // in milliseconds
    float tH2D{ 0.0 }; // in milliseconds
    float tKernel{ 0.0 }; // in milliseconds
    float tD2H{ 0.0 }; // in milliseconds
    float tFrees{ 0.0 }; // in milliseconds

    // Device parameters
    const size_t SIZE_DATA = data.size() * sizeof(T2);
    const size_t SIZE_GRID = gpuGrid.size() * sizeof(T2);
    const size_t SIZE_C = C.size() * sizeof(T2);
    const size_t SIZE_COFFSET = cOffset.size() * sizeof(int);
    const size_t SIZE_IU = iu.size() * sizeof(int);
    const size_t SIZE_IV = iv.size() * sizeof(int);

    T2* dData;
    T2* dGrid;
    T2* dC;
    int* dCOffset;
    int* dIU;
    int* dIV;

    // Allocate device vectors
    hipEventRecord(start);
    hipEventSynchronize(start);
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tAlloc, start, stop);

    hipEventRecord(start);
    hipEventSynchronize(start);
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, gpuGrid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tH2D, start, stop);

    // Kernel launch
    hipEventRecord(start);
    hipEventSynchronize(start);
    typedef hipComplex Complex;
    degridHelper((const Complex*)dGrid, SSIZE, DSIZE, GSIZE, support,
                 (const Complex*)dC, dCOffset, dIU, dIV, (Complex*)dData);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tKernel, start, stop);

    hipEventRecord(start);
    hipEventSynchronize(start);
    hipMemcpy(data.data(), dData, SIZE_DATA, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tD2H, start, stop);

    // Deallocate device vectors
    hipEventRecord(start);
    hipEventSynchronize(start);
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tFrees, start, stop);

    cout << "\nDegridderGPU IN MILLISECONDS:" << endl;
    cout << left << setw(21) << "cudaMallocs"
         << left << setw(21) << "cudaMemcpys (H2D)"
         << left << setw(21) << "kernel"
         << left << setw(21) << "cudaMemcpys (D2H)"
         << left << setw(21) << "frees" << endl;;

    cout << left << setw(21) << tAlloc
         << left << setw(21) << tH2D
         << left << setw(21) << tKernel
         << left << setw(21) << tD2H
         << left << setw(21) << tFrees << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

}

template void DegridderGPU<std::complex<float>>::degridder();
template void DegridderGPU<std::complex<double>>::degridder();
