#include "hip/hip_runtime.h"
#include "DegridderGPU.h"
#include <cmath>
#include "../utilities/MaxError.h"

using std::abs;
using std::cout;
using std::endl;
using std::vector;
using std::complex;

// Error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

void degridHelper(const Complex* dGrid,
    const int SSIZE,
    const int DSIZE,
    const int GSIZE,
    const int support,
    const Complex* dC,
    const int* dCOffset,
    const int* dIU,
    const int* dIV,
    Complex* dData)
{
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    int count = 0;
    //int gridSize = 1024 * devProp.multiProcessorCount; // is starting size, will be reduced as required
    //int gridSize = 256;
    //cout << "Multi processor count: " << devProp.multiProcessorCount << endl;
    //cout << "support: " << support << endl;
    
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    
    for (int dind = 0; dind < DSIZE; dind += GRID_SIZE)
    {
        // if there are less than dimGrid elements left,
        // do the remaining
        //if ((DSIZE - dind) < gridSize)
        //{
        //    gridSize = DSIZE - dind;
        //}

        devDegridKernel<64> << < GRID_SIZE, blockSize >> > (dGrid, GSIZE, dC, dCOffset, dIU, dIV, dData, dind);

        ++count;

        /*
        ++count;
        switch (support)
        {
        case 64:
            devDegridKernel<64> << < gridSize, SSIZE >> > (dGrid, GSIZE, dC, dCOffset, dIU, dIV, dData, dind);
            break;
        default:
            assert(0);
        }
        */
        cudaCheckErrors("cuda kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

}

template <typename T2>
void DegridderGPU<T2>::degridder()
{
    //cout << "\nDegridding on GPU" << endl;

    // Device parameters
    const size_t SIZE_DATA = data.size() * sizeof(T2);
    const size_t SIZE_GRID = gpuGrid.size() * sizeof(T2);
    const size_t SIZE_C = C.size() * sizeof(T2);
    const size_t SIZE_COFFSET = cOffset.size() * sizeof(int);
    const size_t SIZE_IU = iu.size() * sizeof(int);
    const size_t SIZE_IV = iv.size() * sizeof(int);

    T2* dData;
    T2* dGrid;
    T2* dC;
    int* dCOffset;
    int* dIU;
    int* dIV;

    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, gpuGrid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    /*******************************************************************************************************/
    /*******************************************************************************************************/
    // Kernel launch
    typedef hipComplex Complex;

    
    degridHelper((const Complex*)dGrid, SSIZE, DSIZE, GSIZE, support, (const Complex*)dC, dCOffset, dIU, dIV, (Complex*)dData);

    
    /*******************************************************************************************************/
    /*******************************************************************************************************/

    hipMemcpy(data.data(), dData, SIZE_DATA, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");

    

    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
}

template void DegridderGPU<std::complex<float>>::degridder();
template void DegridderGPU<std::complex<double>>::degridder();
