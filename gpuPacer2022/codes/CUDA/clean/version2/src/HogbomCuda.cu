#include "hip/hip_runtime.h"
#include "HogbomCuda.h"

using std::vector;
using std::cout;
using std::endl;

// Error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

struct Peak
{
    size_t pos;
    float val;
};

struct Position 
{
    __host__ __device__
        Position(int _x, int _y) : x(_x), y(_y) { };
    int x;
    int y;
};

__host__ __device__
static Position idxToPos(const size_t idx, const int width)
{
    const int y = idx / width;
    const int x = idx % width;
    return Position(x, y);
}

__host__ __device__
static size_t posToIdx(const int width, const Position& pos)
{
    return (pos.y * width) + pos.x;
}

__global__
void dFindPeak_Step2(float* data, size_t* inIndex, size_t* outIndex, size_t n)
{
    __shared__ float TILE_VAL[BLOCK_SIZE];
    __shared__ size_t TILE_IDX[BLOCK_SIZE];

    size_t tileIdx = threadIdx.x;
    TILE_VAL[tileIdx] = 0.0;
    TILE_IDX[tileIdx] = 0;
    size_t globalIdx = threadIdx.x + blockIdx.x * blockDim.x;

    // grid stride loop to load data
    while (globalIdx < n)
    {
        if (fabs(data[globalIdx]) > fabs(TILE_VAL[tileIdx]))
        {
            TILE_VAL[tileIdx] = data[globalIdx];
            TILE_IDX[tileIdx] = inIndex[globalIdx];
        }
        globalIdx += gridDim.x * blockDim.x;
    }

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();

        // parallel sweep reduction
        if (tileIdx < s)
        {
            if (fabs(TILE_VAL[tileIdx + s]) > fabs(TILE_VAL[tileIdx]))
            {
                TILE_VAL[tileIdx] = TILE_VAL[tileIdx + s];
                TILE_IDX[tileIdx] = TILE_IDX[tileIdx + s];
            }
        }
    }

    if (tileIdx == 0)
    {
        outIndex[blockIdx.x] = TILE_IDX[tileIdx];
        data[blockIdx.x] = TILE_VAL[tileIdx];
    }
}

__global__
void dFindPeak_Step1(const float* data, float* outMax, size_t* outIndex, size_t n)
{
    __shared__ float TILE_VAL[BLOCK_SIZE];
    __shared__ size_t TILE_IDX[BLOCK_SIZE];

    size_t tileIdx = threadIdx.x;

    TILE_VAL[tileIdx] = 0.0;
    TILE_IDX[tileIdx] = 0;

    size_t globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t gridSize = gridDim.x * blockDim.x;

    // grid stride loop to load data
    while (globalIdx < n)
    {
        if (fabs(data[globalIdx]) > fabs(TILE_VAL[tileIdx]))
        {
            TILE_VAL[tileIdx] = data[globalIdx];
            TILE_IDX[tileIdx] = globalIdx;
        }
        globalIdx += gridSize;
    }

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();

        // parallel sweep reduction
        if (tileIdx < s)
        {
            if (fabs(TILE_VAL[tileIdx + s]) > fabs(TILE_VAL[tileIdx]))
            {
                TILE_VAL[tileIdx] = TILE_VAL[tileIdx + s];
                TILE_IDX[tileIdx] = TILE_IDX[tileIdx + s];
            }
        }

    }
    
    if (tileIdx == 0)
    {
        outMax[blockIdx.x] = TILE_VAL[tileIdx];
        outIndex[blockIdx.x] = TILE_IDX[tileIdx];
    }
}

__host__
static Peak findPeak(const float* dData, size_t N)
{
    const size_t SIZE_DATA = N * sizeof(float);
    const size_t SIZE_MAX_VALUE = GRID_SIZE * sizeof(float);
    const size_t SIZE_MAX_INDEX = GRID_SIZE * sizeof(size_t);

    // Host vector for max values
    vector<float> hMax(GRID_SIZE, 0.0);
    // Host vector for index values
    vector<size_t> hIndex(GRID_SIZE, 0);

    // Device vectors
    float* dMax;
    size_t* dIndex;
    size_t* d2Index;

    hipMalloc(&dMax, SIZE_MAX_VALUE);
    hipMalloc(&dIndex, SIZE_MAX_INDEX);
    hipMalloc(&d2Index, sizeof(size_t));
    cudaCheckErrors("hipMalloc failure!");

    dFindPeak_Step1<< <GRID_SIZE, BLOCK_SIZE >> > (dData, dMax, dIndex, N);
    cudaCheckErrors("cuda kernel launch 1 failure!");
    dFindPeak_Step2 << <1, BLOCK_SIZE >> > (dMax, dIndex, d2Index, GRID_SIZE);
    cudaCheckErrors("cuda kernel launch 2 failure!");

    hipMemcpy(hMax.data(), dMax, sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure!");
    hipMemcpy(hIndex.data(), d2Index, sizeof(size_t), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure!");

    Peak p;
    p.val = hMax[0];
    p.pos = hIndex[0];


    hipFree(dMax);
    hipFree(dIndex);
    hipFree(d2Index);
    cudaCheckErrors("hipFree failure!");
    
    return p;
}

__global__
void dSubtractPSF(const float* dPsf,
    const int psfWidth,
    float* dResidual,
    const int residualWidth,
    const int startx, const int starty,
    int const stopx, const int stopy,
    const int diffx, const int diffy,
    const float absPeakVal, const float gain)
{
    const int x = startx + threadIdx.x + (blockIdx.x * blockDim.x);
    const int y = starty + threadIdx.y + (blockIdx.y * blockDim.y);

    // Because thread blocks are of size 16, and the workload is not always
    // a multiple of 16, need to ensure only those threads whose responsibility
    // lies in the work area actually do work
    if (x <= stopx && y <= stopy)
    {
        dResidual[posToIdx(residualWidth, Position(x, y))] -= gain * absPeakVal
            * dPsf[posToIdx(psfWidth, Position(x - diffx, y - diffy))];
    }
}

__host__
static void subtractPSF(const float* dPsf,
    const int psfWidth,
    float* dResidual,
    const int residualWidth,
    const size_t peakPos,
    const size_t psfPeakPos,
    const float absPeakVal,
    const float gain)
{
    const int blockDim = 16;

    const int rx = idxToPos(peakPos, residualWidth).x;
    const int ry = idxToPos(peakPos, residualWidth).y;

    const int px = idxToPos(psfPeakPos, psfWidth).x;
    const int py = idxToPos(psfPeakPos, psfWidth).y;

    const int diffx = rx - px;
    const int diffy = ry - px;

    const int startx = std::max(0, rx - px);
    const int starty = std::max(0, ry - py);

    const int stopx = std::min(residualWidth - 1, rx + (psfWidth - px - 1));
    const int stopy = std::min(residualWidth - 1, ry + (psfWidth - py - 1));

    // Note: Both start* and stop* locations are inclusive.
    const int blocksx = ceil((stopx - startx + 1.0) / static_cast<float>(blockDim));
    const int blocksy = ceil((stopy - starty + 1.0) / static_cast<float>(blockDim));

    dim3 numBlocks(blocksx, blocksy);
    dim3 threadsPerBlock(blockDim, blockDim);
    dSubtractPSF << <numBlocks, threadsPerBlock >> > (dPsf, psfWidth, dResidual, residualWidth,
        startx, starty, stopx, stopy, diffx, diffy, absPeakVal, gain);
    cudaCheckErrors("kernel launch failure in subtractPSF");
}

__host__
void HogbomCuda::deconvolve(const vector<float>& dirty,
    const size_t dirtyWidth,
    const vector<float>& psf,
    const size_t psfWidth,
    vector<float>& model,
    vector<float>& residual)
{
    reportDevice();
    
    const size_t SIZE_DIRTY = dirty.size() * sizeof(float);
    const size_t SIZE_PSF = psf.size() * sizeof(float);
    const size_t SIZE_RESIDUAL = residual.size() * sizeof(float);
    
    residual = dirty;

    // Allocate device memory
    float* dDirty;
    float* dPsf;
    float* dResidual;

    hipMalloc(&dDirty, SIZE_DIRTY);
    hipMalloc(&dPsf, SIZE_PSF);
    hipMalloc(&dResidual, SIZE_RESIDUAL);
    cudaCheckErrors("hipMalloc failure");

    // Copy host to device
    hipMemcpy(dDirty, dirty.data(), SIZE_DIRTY, hipMemcpyHostToDevice);
    hipMemcpy(dPsf, psf.data(), SIZE_PSF, hipMemcpyHostToDevice);
    hipMemcpy(dResidual, residual.data(), SIZE_RESIDUAL, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    // Find peak of psf
    Peak psfPeak = findPeak(dPsf, psf.size());

    cout << "Found peak of PSF: " << "Maximum = " << psfPeak.val
        << " at location " << idxToPos(psfPeak.pos, psfWidth).x << ","
        << idxToPos(psfPeak.pos, psfWidth).y << endl;

    for (unsigned int i = 0; i < gNiters; ++i)
    {
        // Find peak in the residual image
        Peak peak = findPeak(dResidual, residual.size()); 
        
        if ((i + 1) % 100 == 0 || i == 0)
        {
            cout << "Iteration: " << i + 1 << " - Maximum = " << peak.val
                << " at location " << idxToPos(peak.pos, dirtyWidth).x << ","
                << idxToPos(peak.pos, dirtyWidth).y << ", index = " << peak.pos << endl;
        }
        
        // Check if threshold has been reached
        if (abs(peak.val) < gThreshold)
        {
            cout << "Reached stopping threshold" << endl;
            break;
        }

        // Subtract the PSF from the residual image
        // This function will launch a kernel
        // asynchronously, need to sync later
        subtractPSF(dPsf, psfWidth, dResidual, dirtyWidth, peak.pos, psfPeak.pos, peak.val, gGain);

        // Add to model
        model[peak.pos] += peak.val * gGain;
    }

    // Copy device arrays back into the host vector
    hipMemcpy(residual.data(), dResidual, SIZE_RESIDUAL, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");

    hipFree(dDirty);
    hipFree(dPsf);
    hipFree(dResidual);
    cudaCheckErrors("hipFree failure");
}

__host__
void HogbomCuda::reportDevice()
{
    // Report the type of device being used
    int device;
    hipDeviceProp_t devprop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&devprop, device);
    std::cout << "    Using CUDA Device " << device << ": "
        << devprop.name << std::endl;
}