#include "hip/hip_runtime.h"
#include "DegridderGPULessIdle.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

#define MAX_SSIZE 256

__global__
void devDegridKernelLessIdle(
    const Complex* grid,
    const int GSIZE,
    const Complex* C,
    const int support,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* data,
    const int dind)
{

    const int bind = blockIdx.x;
    const int tind = threadIdx.x;

    const int dindLocal = dind + bind;

    const int SSIZE = 2 * support + 1;
    assert(SSIZE == blockDim.x);

    // The actual starting grid point
    __shared__ int gindShared;

    // The Convolution function point from which we offset
    __shared__ int cindShared;

    // Shared memory buffer for the conv pixels in this block (data point)
    __shared__ float sdata_re[MAX_SSIZE];
    __shared__ float sdata_im[MAX_SSIZE];

    if (tind == 0)
    {
        gindShared = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
        cindShared = cOffset[dindLocal];
    }
    __syncthreads();

    Complex original = data[dindLocal];

    for (int row = 0; row < SSIZE; ++row)
    {
        // Make a local copy from shared memory
        int gind = gindShared + GSIZE * row;
        int cind = cindShared + SSIZE * row;

        if (tind < SSIZE)
        {
            const Complex cpix = hipCmulf(grid[gind + tind], C[cind + tind]);
            sdata_re[tind] = cpix.x;
            sdata_im[tind] = cpix.y;
            __syncthreads();

            for (unsigned int s = SSIZE / 2; s > 0; s /= 2)
            {
                // reduce the real part with threads 0:SSIZE/2
                if ((tind < s) && (tind + s < SSIZE)) {
                    sdata_re[tind] += sdata_re[tind + s];
                }
                // reduce the imaginary part with threads SSIZE/2:SSIZE
                if ((tind > SSIZE - 1 - s) && (tind - s >= 0)) {
                    sdata_im[tind] += sdata_im[tind - s];
                }
                __syncthreads();
            }
            // because SSIZE is odd, the real accumulation ends in the first thread but misses the last thread
            // while the imaginary accumulation ends in the last thread but misses the first thread.
            // So add the last to the first before moving on
            if (tind == 0)
            {
                sdata_re[tind] += sdata_re[SSIZE - 1];
                sdata_im[tind] += sdata_im[SSIZE - 1];
            }
            __syncthreads();

        }

        if (tind == 0)
        {
            original = hipCaddf(original, make_hipComplex(sdata_re[tind], sdata_im[tind]));
        }
    }

    if (tind == 0)
    {
        data[dindLocal] = original;
    }

}

template<typename T2>
void DegridderGPULessIdle<T2>::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    gpuCheckErrors("hipMalloc failure");
}

template<typename T2>
void DegridderGPULessIdle<T2>::copyH2D()
{
    hipMemcpy(dData, this->data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, this->grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, this->C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, this->cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, this->iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, this->iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    gpuCheckErrors("hipMemcpy H2D failure");
}

template<typename T2>
DegridderGPULessIdle<T2>::~DegridderGPULessIdle()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    gpuCheckErrors("hipFree failure");
}

template <typename T2>
void DegridderGPULessIdle<T2>::degridder()
{
    deviceAllocations();
    copyH2D();

    // Kernel launch
    const size_t DSIZE = this->data.size();
    typedef hipComplex Complex;

    const int SSIZE = 2 * this->support + 1;

    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), hipFuncCachePreferL1);

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (this->support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devDegridKernelLessIdle <<<gridSize, SSIZE>>> ((const Complex*)dGrid, GSIZE, (const Complex*)dC, this->support, dCOffset, dIU, dIV, (Complex*)dData, dind);

        gpuCheckErrors("hip kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(this->data.data(), dData, SIZE_DATA, hipMemcpyDeviceToHost);
    gpuCheckErrors("hipMemcpy D2H failure");
}

template void DegridderGPULessIdle<std::complex<float>>::degridder();
template void DegridderGPULessIdle<std::complex<double>>::degridder();
template void DegridderGPULessIdle<std::complex<float>>::deviceAllocations();
template void DegridderGPULessIdle<std::complex<double>>::deviceAllocations();
template void DegridderGPULessIdle<std::complex<float>>::copyH2D();
template void DegridderGPULessIdle<std::complex<double>>::copyH2D();
template DegridderGPULessIdle<std::complex<float>>::~DegridderGPULessIdle();
template DegridderGPULessIdle<std::complex<double>>::~DegridderGPULessIdle();
