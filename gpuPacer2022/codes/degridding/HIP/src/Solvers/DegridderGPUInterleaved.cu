#include "hip/hip_runtime.h"
#include "DegridderGPUInterleaved.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

#define MAX_SSIZE 256

__global__
void devDegridKernelInterleaved(
    const Complex* grid,
    const int GSIZE,
    const Complex* C,
    const int support,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* data,
    const int dind)
{

    const int bind = blockIdx.x;
    const int tind = threadIdx.x;

    const int dindLocal = dind + bind;

    const int SSIZE = 2 * support + 1;
    assert(SSIZE == blockDim.x);

    // The actual starting grid point
    __shared__ int gindShared;

    // The Convolution function point from which we offset
    __shared__ int cindShared;

    // Shared memory buffer for the conv pixels in this block (data point)
    __shared__ float sdata_re[MAX_SSIZE];
    __shared__ float sdata_im[MAX_SSIZE];

    if (tind == 0)
    {
        gindShared = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
        cindShared = cOffset[dindLocal];
    }
    __syncthreads();

    Complex original = data[dindLocal];

    for (int row = 0; row < SSIZE; ++row)
    {
        // Make a local copy from shared memory
        int gind = gindShared + GSIZE * row;
        int cind = cindShared + SSIZE * row;

        if (tind < SSIZE)
        {
            const Complex cpix = hipCmulf(grid[gind + tind], C[cind + tind]);
            sdata_re[tind] = cpix.x;
            sdata_im[tind] = cpix.y;
            __syncthreads();

            for (unsigned int s = 1; s < SSIZE; s *= 2)
            {
                int index = 2 * s * tind;
                if (index + s < SSIZE) 
                {
                    //sdata[tind] = hipCaddf(sdata[tind], sdata[tind + s]);
                    sdata_re[index] += sdata_re[index + s];
                    sdata_im[index] += sdata_im[index + s];
                }
                __syncthreads();
            }

        }

        if (tind == 0)
        {
            original = hipCaddf(original, make_hipComplex(sdata_re[tind], sdata_im[tind]));
        }
    }

    if (tind == 0)
    {
        data[dindLocal] = original;
    }

}

void DegridderGPUInterleaved::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    gpuCheckErrors("hipMalloc failure");
}

void DegridderGPUInterleaved::copyH2D()
{
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    gpuCheckErrors("hipMemcpy H2D failure");
}

DegridderGPUInterleaved::~DegridderGPUInterleaved()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    gpuCheckErrors("hipFree failure");
}

void DegridderGPUInterleaved::degridder()
{
    deviceAllocations();
    copyH2D();

    // Kernel launch
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;

    const int SSIZE = 2 * support + 1;

    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), hipFuncCachePreferL1);

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devDegridKernelInterleaved <<<gridSize, SSIZE>>> ((const Complex*)dGrid, GSIZE, (const Complex*)dC, support, dCOffset, dIU, dIV, (Complex*)dData, dind);

        gpuCheckErrors("hip kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(data.data(), dData, SIZE_DATA, hipMemcpyDeviceToHost);
    gpuCheckErrors("hipMemcpy D2H failure");
}

