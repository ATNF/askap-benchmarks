#include "hip/hip_runtime.h"
#include "WarmupGPU.h"

using std::vector;
using std::cout;
using std::endl;

__global__
void vectorAdd(const float* a, const float* b, float* c, const size_t N)
{
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
	{
		for (int j = 0; j < 250; ++j)
		{
			c[i] = a[i] + b[i];
		}
	}
}

void WarmupGPU::warmup() const
{
	vector<float> a(N, 1.0);
	vector<float> b(N, 2.0);
	vector<float> c(N, 0.0);
	vector<float> cAnswer(N, 3.0);

	const size_t SIZE = N * sizeof(float);

	float* dA;
	float* dB;
	float* dC;

	gpuMalloc(&dA, SIZE);
	gpuMalloc(&dB, SIZE);
	gpuMalloc(&dC, SIZE);

	gpuMemcpy(dA, a.data(), SIZE, gpuMemcpyHostToDevice);
	gpuMemcpy(dB, b.data(), SIZE, gpuMemcpyHostToDevice);

	const int blockSize = 1024;
	const int gridSize = N / 1024;

	vectorAdd<<<gridSize, blockSize>>> (dA, dB, dC, N);

	gpuMemcpy(c.data(), dC, SIZE, gpuMemcpyDeviceToHost);

	MaxError<float> maximumError;
	LocalLog() << "Verifying warmup launch" << endl;
	maximumError.maxError(c, cAnswer);

	gpuFree(dA);
	gpuFree(dB);
	gpuFree(dC);
}
