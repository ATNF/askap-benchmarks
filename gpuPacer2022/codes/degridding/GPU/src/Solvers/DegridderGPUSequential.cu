#include "hip/hip_runtime.h"
#include "DegridderGPUSequential.h"
#include "../../utilities/gpuCommon.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;


__global__
void devDegridKernelSequential(
    const Complex* grid,
    const int GSIZE,
    const Complex* C,
    const int support,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* data,
    const int dind)
{

    const int bind = blockIdx.x;
    const int tind = threadIdx.x;

    const int dindLocal = dind + bind;

    const int SSIZE = 2 * support + 1;
    assert(SSIZE == blockDim.x);

    // The actual starting grid point
    __shared__ int gindShared;

    // The Convolution function point from which we offset
    __shared__ int cindShared;

    // Shared memory buffer for the conv pixels in this block (data point)
    __shared__ float sdata_re[MAX_SSIZE];
    __shared__ float sdata_im[MAX_SSIZE];

    if (tind == 0)
    {
        gindShared = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
        cindShared = cOffset[dindLocal];
    }
    __syncthreads();

    Complex original = data[dindLocal];

    for (int row = 0; row < SSIZE; ++row)
    {
        // Make a local copy from shared memory
        int gind = gindShared + GSIZE * row;
        int cind = cindShared + SSIZE * row;

        if (tind < SSIZE)
        {
            const Complex cpix = gpuCmulf(grid[gind + tind], C[cind + tind]);
            sdata_re[tind] = cpix.x;
            sdata_im[tind] = cpix.y;
            __syncthreads();

            for (unsigned int s = SSIZE / 2; s > 0; s /= 2)
            {
                if ((tind < s) && (tind + s < SSIZE)) 
                {
                    //sdata[tind] = gpuCaddf(sdata[tind], sdata[tind + s]);
                    sdata_re[tind] += sdata_re[tind + s];
                    sdata_im[tind] += sdata_im[tind + s];
                }
                __syncthreads();
            }
            // because SSIZE is odd, reduction #3 misses the last thread
            if (tind == 0)
            {
                sdata_re[tind] += sdata_re[SSIZE - 1];
                sdata_im[tind] += sdata_im[SSIZE - 1];
            }
            __syncthreads();

        }

        if (tind == 0)
        {
            original = gpuCaddf(original, make_gpuComplex(sdata_re[tind], sdata_im[tind]));
        }
    }

    if (tind == 0)
    {
        data[dindLocal] = original;
    }

}

template<typename T2>
void DegridderGPUSequential<T2>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dData, SIZE_DATA);
    gpuMalloc(&dGrid, SIZE_GRID);
    gpuMalloc(&dC, SIZE_C);
    gpuMalloc(&dCOffset, SIZE_COFFSET);
    gpuMalloc(&dIU, SIZE_IU);
    gpuMalloc(&dIV, SIZE_IV);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T2>
void DegridderGPUSequential<T2>::copyH2D()
{
    gpuMemcpy(dData, this->data.data(), SIZE_DATA, gpuMemcpyHostToDevice);
    gpuMemcpy(dGrid, this->grid.data(), SIZE_GRID, gpuMemcpyHostToDevice);
    gpuMemcpy(dC, this->C.data(), SIZE_C, gpuMemcpyHostToDevice);
    gpuMemcpy(dCOffset, this->cOffset.data(), SIZE_COFFSET, gpuMemcpyHostToDevice);
    gpuMemcpy(dIU, this->iu.data(), SIZE_IU, gpuMemcpyHostToDevice);
    gpuMemcpy(dIV, this->iv.data(), SIZE_IV, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T2>
DegridderGPUSequential<T2>::~DegridderGPUSequential()
{
    // Deallocate device vectors
    gpuFree(dData);
    gpuFree(dGrid);
    gpuFree(dC);
    gpuFree(dCOffset);
    gpuFree(dIU);
    gpuFree(dIV);
    gpuCheckErrors("gpuFree failure");
}

template <typename T2>
void DegridderGPUSequential<T2>::degridder()
{
    deviceAllocations();
    copyH2D();

    // Kernel launch
    const size_t DSIZE = this->data.size();

    const int SSIZE = 2 * this->support + 1;

    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), hipFuncCachePreferL1);

    int device;
    gpuGetDevice(&device);
    gpuDeviceProp_t devProp;
    gpuGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (this->support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devDegridKernelSequential <<<gridSize, SSIZE>>> ((const Complex*)dGrid, GSIZE, (const Complex*)dC, this->support, dCOffset, dIU, dIV, (Complex*)dData, dind);

        gpuCheckErrors("gpu kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    gpuMemcpy(this->data.data(), dData, SIZE_DATA, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template void DegridderGPUSequential<std::complex<float>>::degridder();
template void DegridderGPUSequential<std::complex<double>>::degridder();
template void DegridderGPUSequential<std::complex<float>>::deviceAllocations();
template void DegridderGPUSequential<std::complex<double>>::deviceAllocations();
template void DegridderGPUSequential<std::complex<float>>::copyH2D();
template void DegridderGPUSequential<std::complex<double>>::copyH2D();
template DegridderGPUSequential<std::complex<float>>::~DegridderGPUSequential();
template DegridderGPUSequential<std::complex<double>>::~DegridderGPUSequential();
