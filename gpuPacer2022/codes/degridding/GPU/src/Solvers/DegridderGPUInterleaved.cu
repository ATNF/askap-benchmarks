#include "hip/hip_runtime.h"
#include "DegridderGPUInterleaved.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

#define MAX_SSIZE 256

__global__
void devDegridKernelInterleaved(
    const Complex* grid,
    const int GSIZE,
    const Complex* C,
    const int support,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* data,
    const int dind)
{

    const int bind = blockIdx.x;
    const int tind = threadIdx.x;

    const int dindLocal = dind + bind;

    const int SSIZE = 2 * support + 1;
    assert(SSIZE == blockDim.x);

    // The actual starting grid point
    __shared__ int gindShared;

    // The Convolution function point from which we offset
    __shared__ int cindShared;

    // Shared memory buffer for the conv pixels in this block (data point)
    __shared__ float sdata_re[MAX_SSIZE];
    __shared__ float sdata_im[MAX_SSIZE];

    if (tind == 0)
    {
        gindShared = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
        cindShared = cOffset[dindLocal];
    }
    __syncthreads();

    Complex original = data[dindLocal];

    for (int row = 0; row < SSIZE; ++row)
    {
        // Make a local copy from shared memory
        int gind = gindShared + GSIZE * row;
        int cind = cindShared + SSIZE * row;

        if (tind < SSIZE)
        {
            const Complex cpix = gpuCmulf(grid[gind + tind], C[cind + tind]);
            sdata_re[tind] = cpix.x;
            sdata_im[tind] = cpix.y;
            __syncthreads();

            for (unsigned int s = 1; s < SSIZE; s *= 2)
            {
                int index = 2 * s * tind;
                if (index + s < SSIZE) 
                {
                    //sdata[tind] = gpuCaddf(sdata[tind], sdata[tind + s]);
                    sdata_re[index] += sdata_re[index + s];
                    sdata_im[index] += sdata_im[index + s];
                }
                __syncthreads();
            }

        }

        if (tind == 0)
        {
            original = gpuCaddf(original, make_gpuComplex(sdata_re[tind], sdata_im[tind]));
        }
    }

    if (tind == 0)
    {
        data[dindLocal] = original;
    }

}

template<typename T2>
void DegridderGPUInterleaved<T2>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dData, SIZE_DATA);
    gpuMalloc(&dGrid, SIZE_GRID);
    gpuMalloc(&dC, SIZE_C);
    gpuMalloc(&dCOffset, SIZE_COFFSET);
    gpuMalloc(&dIU, SIZE_IU);
    gpuMalloc(&dIV, SIZE_IV);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T2>
void DegridderGPUInterleaved<T2>::copyH2D()
{
    gpuMemcpy(dData, this->data.data(), SIZE_DATA, gpuMemcpyHostToDevice);
    gpuMemcpy(dGrid, this->grid.data(), SIZE_GRID, gpuMemcpyHostToDevice);
    gpuMemcpy(dC, this->C.data(), SIZE_C, gpuMemcpyHostToDevice);
    gpuMemcpy(dCOffset, this->cOffset.data(), SIZE_COFFSET, gpuMemcpyHostToDevice);
    gpuMemcpy(dIU, this->iu.data(), SIZE_IU, gpuMemcpyHostToDevice);
    gpuMemcpy(dIV, this->iv.data(), SIZE_IV, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T2>
DegridderGPUInterleaved<T2>::~DegridderGPUInterleaved()
{
    // Deallocate device vectors
    gpuFree(dData);
    gpuFree(dGrid);
    gpuFree(dC);
    gpuFree(dCOffset);
    gpuFree(dIU);
    gpuFree(dIV);
    gpuCheckErrors("gpuFree failure");
}

template <typename T2>
void DegridderGPUInterleaved<T2>::degridder()
{
    deviceAllocations();
    copyH2D();

    // Kernel launch
    const size_t DSIZE = this->data.size();
    //typedef gpuComplex Complex;

    const int SSIZE = 2 * this->support + 1;

    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), hipFuncCachePreferL1);

    int device;
    gpuGetDevice(&device);
    gpuDeviceProp_t devProp;
    gpuGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (this->support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devDegridKernelInterleaved <<<gridSize, SSIZE>>> ((const Complex*)dGrid, GSIZE, (const Complex*)dC, this->support, dCOffset, dIU, dIV, (Complex*)dData, dind);

        gpuCheckErrors("gpu kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    gpuMemcpy(this->data.data(), dData, SIZE_DATA, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template void DegridderGPUInterleaved<std::complex<float>>::degridder();
template void DegridderGPUInterleaved<std::complex<double>>::degridder();
template void DegridderGPUInterleaved<std::complex<float>>::deviceAllocations();
template void DegridderGPUInterleaved<std::complex<double>>::deviceAllocations();
template void DegridderGPUInterleaved<std::complex<float>>::copyH2D();
template void DegridderGPUInterleaved<std::complex<double>>::copyH2D();
template DegridderGPUInterleaved<std::complex<float>>::~DegridderGPUInterleaved();
template DegridderGPUInterleaved<std::complex<double>>::~DegridderGPUInterleaved();