#include "hip/hip_runtime.h"
#include "DegridderGPUWarpShuffle.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;
#define FULL_MASK 0xffffffff

#ifdef __NVCC__
#define WARPSIZE 32
#else
#define WARPSIZE 64
#endif

const int supportWS = 64;

// launch_bounds__(2*support+1, 8)
__global__
void devDegridKernelWarpShuffle(
    const Complex* grid,
    const int GSIZE,
    const Complex* C,
    const int support,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* data,
    const int dind)
{

    int dindLocal = dind + blockIdx.x;
    int gindStart = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
    int cindStart = cOffset[dindLocal];
    int SSIZE = 2 * support + 1;
    int suppu = threadIdx.x;

    Complex dOrig = data[dindLocal];
    // suppv loop
    for (int suppv = 0; suppv < SSIZE; ++suppv)
    {
        int gind = gindStart + GSIZE * suppv;
        int cind = cindStart + SSIZE * suppv;
        Complex sum = hipCmulf(grid[gind + suppu], C[cind + suppu]);

        __syncthreads();
        // Reduce within each warp
        if (suppu < SSIZE)
        {
            for (int offset = WARPSIZE / 2; offset > 0; offset /= 2)
            {
#ifdef __NVCC__		
                sum.x += __shfl_down_sync(FULL_MASK, sum.x, offset, WARPSIZE);
                sum.y += __shfl_down_sync(FULL_MASK, sum.y, offset, WARPSIZE);
#else	  
                sum.x += __shfl_down(sum.x, offset, WARPSIZE);
                sum.y += __shfl_down(sum.y, offset, WARPSIZE);
#endif	  
            }

        }

        // Gather warp sums into shared memory
        const int NUMWARPS = (2 * supportWS + 1) / WARPSIZE + 1;
        __shared__ Complex dataShared[NUMWARPS];

        int warp = suppu / WARPSIZE;
        int lane = threadIdx.x & (WARPSIZE - 1); // the lead thread in the warp

        if (lane == 0)
        {
            dataShared[warp] = sum;
        }

        __syncthreads();
        // combine warp sums using a single thread in this block
        if (suppu == 0)
        {
            for (int w = 1; w < NUMWARPS; w++)
            {
                sum = hipCaddf(sum, dataShared[w]);
            }

            dOrig = hipCaddf(dOrig, sum);
        }
    }
    if (suppu == 0)
    {
        data[dindLocal] = dOrig;
    }
}

void DegridderGPUWarpShuffle::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");
}

void DegridderGPUWarpShuffle::copyH2D()
{
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
}

DegridderGPUWarpShuffle::~DegridderGPUWarpShuffle()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
}

void DegridderGPUWarpShuffle::degridder()
{
    deviceAllocations();
    copyH2D();

    // Kernel launch
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;

    const int SSIZE = 2 * support + 1;

    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), hipFuncCachePreferL1);

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    //int gridSize = 1024 * devProp.multiProcessorCount; // is starting size, will be reduced as required
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left,
        // do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }
        devDegridKernelWarpShuffle <<< gridSize, SSIZE >>> ((const Complex*)dGrid, GSIZE, (const Complex*)dC, support, dCOffset, dIU, dIV, (Complex*)dData, dind);
        cudaCheckErrors("cuda kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(data.data(), dData, SIZE_DATA, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
}

