#include "hip/hip_runtime.h"
#include "DegridderGPUSequential.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

#define MAX_SSIZE 256

__global__
void devDegridKernelSequential(
    const Complex* grid,
    const int GSIZE,
    const Complex* C,
    const int support,
    const int* cOffset,
    const int* iu,
    const int* iv,
    Complex* data,
    const int dind)
{

    const int bind = blockIdx.x;
    const int tind = threadIdx.x;

    const int dindLocal = dind + bind;

    const int SSIZE = 2 * support + 1;
    assert(SSIZE == blockDim.x);

    // The actual starting grid point
    __shared__ int gindShared;

    // The Convolution function point from which we offset
    __shared__ int cindShared;

    // Shared memory buffer for the conv pixels in this block (data point)
    __shared__ float sdata_re[MAX_SSIZE];
    __shared__ float sdata_im[MAX_SSIZE];

    if (tind == 0)
    {
        gindShared = iu[dindLocal] + GSIZE * iv[dindLocal] - support;
        cindShared = cOffset[dindLocal];
    }
    __syncthreads();

    Complex original = data[dindLocal];

    for (int row = 0; row < SSIZE; ++row)
    {
        // Make a local copy from shared memory
        int gind = gindShared + GSIZE * row;
        int cind = cindShared + SSIZE * row;

        if (tind < SSIZE)
        {
            const Complex cpix = hipCmulf(grid[gind + tind], C[cind + tind]);
            sdata_re[tind] = cpix.x;
            sdata_im[tind] = cpix.y;
            __syncthreads();

            for (unsigned int s = SSIZE / 2; s > 0; s /= 2)
            {
                if ((tind < s) && (tind + s < SSIZE)) 
                {
                    //sdata[tind] = hipCaddf(sdata[tind], sdata[tind + s]);
                    sdata_re[tind] += sdata_re[tind + s];
                    sdata_im[tind] += sdata_im[tind + s];
                }
                __syncthreads();
            }
            // because SSIZE is odd, reduction #3 misses the last thread
            if (tind == 0)
            {
                sdata_re[tind] += sdata_re[SSIZE - 1];
                sdata_im[tind] += sdata_im[SSIZE - 1];
            }
            __syncthreads();

        }

        if (tind == 0)
        {
            original = hipCaddf(original, make_hipComplex(sdata_re[tind], sdata_im[tind]));
        }
    }

    if (tind == 0)
    {
        data[dindLocal] = original;
    }

}

template<typename T2>
void DegridderGPUSequential<T2>::deviceAllocations()
{
    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");
}

template<typename T2>
void DegridderGPUSequential<T2>::copyH2D()
{
    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, grid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
}

template<typename T2>
DegridderGPUSequential<T2>::~DegridderGPUSequential()
{
    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
}

template <typename T2>
void DegridderGPUSequential<T2>::degridder()
{
    deviceAllocations();
    copyH2D();

    // Kernel launch
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;

    const int SSIZE = 2 * support + 1;

    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernelOlder), hipFuncCachePreferL1);

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);

    int gridSize = devProp.maxGridSize[0] / (support + 1);  // launch kernels for this number of samples at a time
    assert(SSIZE <= devProp.maxThreadsPerBlock);

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += gridSize)
    {
        // if there are less than dimGrid elements left, do the remaining
        if ((DSIZE - dind) < gridSize)
        {
            gridSize = DSIZE - dind;
        }

        ++count;

        devDegridKernelSequential << < gridSize, SSIZE >> > ((const Complex*)dGrid, GSIZE, (const Complex*)dC, support, dCOffset, dIU, dIV, (Complex*)dData, dind);

        cudaCheckErrors("cuda kernel launch failure");
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(data.data(), dData, SIZE_DATA, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");
}

template void DegridderGPUSequential<std::complex<float>>::degridder();
template void DegridderGPUSequential<std::complex<double>>::degridder();
template void DegridderGPUSequential<std::complex<float>>::deviceAllocations();
template void DegridderGPUSequential<std::complex<double>>::deviceAllocations();
template void DegridderGPUSequential<std::complex<float>>::copyH2D();
template void DegridderGPUSequential<std::complex<double>>::copyH2D();
template DegridderGPUSequential<std::complex<float>>::~DegridderGPUSequential();
template DegridderGPUSequential<std::complex<double>>::~DegridderGPUSequential();
