#include "hip/hip_runtime.h"
#include "GridderGPU.h"

using std::cout;
using std::endl;
using std::vector;
using std::complex;

// Error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

int gridStep(const int DSIZE, const int SSIZE, const int dind, const std::vector<int>&iu, const std::vector<int>&iv);

template <typename T2>
void GridderGPU<T2>::gridder()
{
    cout << "\nGridding on GPU" << endl;

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    cudaCheckErrors("cudaEvent create failure");

    // Device parameters
    const size_t SIZE_DATA = data.size() * sizeof(T2);
    const size_t SIZE_GRID = gpuGrid.size() * sizeof(T2);
    const size_t SIZE_C = C.size() * sizeof(T2);
    const size_t SIZE_COFFSET = cOffset.size() * sizeof(int);
    const size_t SIZE_IU = iu.size() * sizeof(int);
    const size_t SIZE_IV = iv.size() * sizeof(int);

    T2* dData;
    T2* dGrid;
    T2* dC;
    int* dCOffset;
    int* dIU;
    int* dIV;

    // Allocate device vectors
    hipMalloc(&dData, SIZE_DATA);
    hipMalloc(&dGrid, SIZE_GRID);
    hipMalloc(&dC, SIZE_C);
    hipMalloc(&dCOffset, SIZE_COFFSET);
    hipMalloc(&dIU, SIZE_IU);
    hipMalloc(&dIV, SIZE_IV);
    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(dData, data.data(), SIZE_DATA, hipMemcpyHostToDevice);
    hipMemcpy(dGrid, gpuGrid.data(), SIZE_GRID, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), SIZE_C, hipMemcpyHostToDevice);
    hipMemcpy(dCOffset, cOffset.data(), SIZE_COFFSET, hipMemcpyHostToDevice);
    hipMemcpy(dIU, iu.data(), SIZE_IU, hipMemcpyHostToDevice);
    hipMemcpy(dIV, iv.data(), SIZE_IV, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    /*******************************************************************************************************/
    /*******************************************************************************************************/
    // Kernel launch
    cout << "Kernel launch" << endl;
    const size_t DSIZE = data.size();
    typedef hipComplex Complex;
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(devGridKernel), hipFuncCachePreferL1);

    const int SSIZE = 2 * support + 1;
    int step = 1;

    /*
    This loop steps through each spectral sample
    either 1 or 2 at a time. It will do multiple samples
    if the regions involved do not overlap. If they do,
    only the non-overlapping samples are gridded.

    Gridding multiple points is better, because giving the
    GPU more work to do allows it to hide memory latency
    better. The call to d_gridKernel() is asynchronous
    so subsequent calls to gridStep() overlap with the actual gridding.
    */

    int count = 0;
    for (int dind = 0; dind < DSIZE; dind += step)
    {
        step = gridStep(DSIZE, SSIZE, dind, iu, iv);
        cout << "Step = " << step << endl;
        dim3 gridDim(SSIZE, step);
        devGridKernel << <gridDim, SSIZE >> > ((const Complex*)dData, support, (const Complex*)dC, dCOffset, dIU, dIV, (Complex*)dGrid, GSIZE, dind);
        cudaCheckErrors("kernel launch (devGridKernel_v0) failure");
        count++;
    }
    cout << "Used " << count << " kernel launches." << endl;

    hipMemcpy(gpuGrid.data(), dGrid, SIZE_GRID, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");

    // Deallocate device vectors
    hipFree(dData);
    hipFree(dGrid);
    hipFree(dC);
    hipFree(dCOffset);
    hipFree(dIU);
    hipFree(dIV);
    cudaCheckErrors("hipFree failure");
}

int gridStep(const int DSIZE, const int SSIZE, const int dind, const std::vector<int>& iu, const std::vector<int>& iv)
{
    const int MAXSAMPLES = 32;
    for (int step = 1; step <= MAXSAMPLES; ++step)
    {
        for (int check = (step - 1); check >= 0; --check)
        {
            if (!((dind + step) < DSIZE && (
                abs(iu[dind + step] - iu[dind + check]) > SSIZE ||
                abs(iv[dind + step] - iv[dind + check]) > SSIZE)
                ))
            {
                return step;
            }
        }
    }
    return MAXSAMPLES;
}

template void GridderGPU<std::complex<float>>::gridder();
template void GridderGPU<std::complex<double>>::gridder();

